// gpu_kernels.cu - 4 CUDA Kernels for Fast DDS

#include <hip/hip_runtime.h>

#include <stdio.h>

#define BLOCK_SIZE 256

// Kernel 1: Vector Addition
__global__ void vectorAddKernel(const float* a, const float* b, float* c, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

// Kernel 2: FFT Transform (simplified)
__global__ void fftTransformKernel(const float* input, float* output, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        float angle = -2.0f * M_PI * idx / n;
        output[idx] = input[idx] * cosf(angle) + input[idx] * sinf(angle);
    }
}

// Kernel 3: 1D Convolution
__global__ void convolution1DKernel(const float* input, const float* kernel, 
                                    float* output, int n, int kernel_size) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (idx < n) {
        float sum = 0.0f;
        int half_kernel = kernel_size / 2;
        
        for (int k = -half_kernel; k <= half_kernel; k++) {
            int input_idx = idx + k;
            if (input_idx >= 0 && input_idx < n) {
                sum += input[input_idx] * kernel[k + half_kernel];
            }
        }
        output[idx] = sum;
    }
}

// Kernel 4: Matrix Multiplication (square matrices)
__global__ void matrixMultKernel(const float* A, const float* B, float* C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < width && col < width) {
        float sum = 0.0f;
        for (int k = 0; k < width; k++) {
            sum += A[row * width + k] * B[k * width + col];
        }
        C[row * width + col] = sum;
    }
}

// C++ Interface Functions
extern "C" {
    
    // Initialize CUDA
    int initCuda() {
        int deviceCount;
        hipGetDeviceCount(&deviceCount);
        
        if (deviceCount == 0) {
            printf("No CUDA devices found!\n");
            return -1;
        }
        
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        printf("GPU: %s\n", prop.name);
        printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
        printf("Memory: %.2f GB\n", prop.totalGlobalMem / (1024.0 * 1024.0 * 1024.0));
        
        return 0;
    }
    
    // Execute Kernel 1: Vector Add
    float executeVectorAdd(const float* h_a, const float* h_b, float* h_c, int n) {
        float *d_a, *d_b, *d_c;
        size_t size = n * sizeof(float);
        
        // Allocate GPU memory
        hipMalloc(&d_a, size);
        hipMalloc(&d_b, size);
        hipMalloc(&d_c, size);
        
        // Create events for timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        // Copy data to GPU
        hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
        
        // Execute kernel
        int gridSize = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
        
        hipEventRecord(start);
        vectorAddKernel<<<gridSize, BLOCK_SIZE>>>(d_a, d_b, d_c, n);
        hipEventRecord(stop);
        
        // Copy result back
        hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
        
        // Calculate time
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        
        // Cleanup
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        
        return milliseconds;
    }
    
    // Execute Kernel 2: FFT Transform
    float executeFftTransform(const float* h_input, float* h_output, int n) {
        float *d_input, *d_output;
        size_t size = n * sizeof(float);
        
        hipMalloc(&d_input, size);
        hipMalloc(&d_output, size);
        
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);
        
        int gridSize = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
        
        hipEventRecord(start);
        fftTransformKernel<<<gridSize, BLOCK_SIZE>>>(d_input, d_output, n);
        hipEventRecord(stop);
        
        hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);
        
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        
        hipFree(d_input);
        hipFree(d_output);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        
        return milliseconds;
    }
    
    // Execute Kernel 3: Convolution
    float executeConvolution(const float* h_input, const float* h_kernel, 
                            float* h_output, int n, int kernel_size) {
        float *d_input, *d_kernel, *d_output;
        
        hipMalloc(&d_input, n * sizeof(float));
        hipMalloc(&d_kernel, kernel_size * sizeof(float));
        hipMalloc(&d_output, n * sizeof(float));
        
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        hipMemcpy(d_input, h_input, n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_kernel, h_kernel, kernel_size * sizeof(float), hipMemcpyHostToDevice);
        
        int gridSize = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
        
        hipEventRecord(start);
        convolution1DKernel<<<gridSize, BLOCK_SIZE>>>(d_input, d_kernel, d_output, n, kernel_size);
        hipEventRecord(stop);
        
        hipMemcpy(h_output, d_output, n * sizeof(float), hipMemcpyDeviceToHost);
        
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        
        hipFree(d_input);
        hipFree(d_kernel);
        hipFree(d_output);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        
        return milliseconds;
    }
    
    // Execute Kernel 4: Matrix Multiplication
    float executeMatrixMult(const float* h_A, const float* h_B, float* h_C, int width) {
        float *d_A, *d_B, *d_C;
        size_t size = width * width * sizeof(float);
        
        hipMalloc(&d_A, size);
        hipMalloc(&d_B, size);
        hipMalloc(&d_C, size);
        
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
        
        dim3 block(16, 16);
        dim3 grid((width + block.x - 1) / block.x, (width + block.y - 1) / block.y);
        
        hipEventRecord(start);
        matrixMultKernel<<<grid, block>>>(d_A, d_B, d_C, width);
        hipEventRecord(stop);
        
        hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
        
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        
        return milliseconds;
    }
}
